
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sumParallel(int *dev_sum, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid <= num)
    {

	/* se usa la función atomicAdd para realizar la sumatoria y  garantizar que varios hilos no vayan a escribir simultáneamente en la misma ubicación de memoria. */
        atomicAdd(dev_sum, tid);
        tid += stride; //coge el identificado unico de cada hilo y le agrega el paso para pasar al siguiente hilo
    }
}

int main()
{
    int num, sum = 0;
    int *dev_sum; //referencia de la variable que se ubicara en el device

     /*Inicializamos las variables con las cuales tomaremos el tiempo */
    hipEvent_t start, stop;
    float elapsedTime;
   
    printf("Enter a positive integer: ");
    scanf("%d", &num);

    hipMalloc((void**)&dev_sum, sizeof(int)); //reservamos espacio de memoria
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice); //copaimos la variable desde el host al sum

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); //comienza a tomar el tiempo

    sumParallel<<<gridSize, blockSize>>>(dev_sum, num); //invocamos el kernel sumParallel que es el que se encarga de realizar la suma
    
    hipEventRecord(stop, 0); //para de tomar el tiempo
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost); //copiamos el resultado ahora en sentido contrario, es decir desde el device hasta el host
    hipFree(dev_sum); //liberamos la memoria reservada

    printf("\nSum = %d\n", sum); //imprimimos el resultado de la suma

    printf("Elapsed Time: %.6f segundos\n", elapsedTime/1000); //me imprime el tiempo que demoro

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
