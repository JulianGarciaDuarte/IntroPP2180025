#include "hip/hip_runtime.h"
include <stdio.h>

__global__ void sumaSimple(int n, int *result) {
    int tid=threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int partialSum = 0;

    for (int i = tid + 1; i <= n; i += stride){
     	partialSum += 1;
    }

    atomicAdd(result, partialSum);
}

int main() {
    float hostResult = 0;

    // Sets the size of the numbers
    int n = 100000;

    // Declara y ubica la memoria del lado del host
    int *devResult;
    hipMalloc((void**)&devResult, sizeof(int));
    
    // Copia los datos de entrada del host al dispositivo
    hipMemcpy(devResult, &hostResult, sizeof(int), hipMemcpyHostToDevice);
    
    // Set block and grid dimentions
    int blocksize=256;
    int gridSize = (n + blockSize - 1)/blockSize;
    
    // Lanza el kernel
    sumaSimple<<<gridSize, blockSize>>>(n, devResult);

    // Copia el resultado de regreso a la memoria del host
    hipMemcpy(&hostResult, devResult, sizeof(int), hipMemcpyDeviceToHost);
    
    // Libera la memoria del dispositivo
    hipFree(devResult);
    
    // Muestra el resultado
    printf("La suma de los primeros %d numeros naturales is %d/n", n, hostResult);
    return 0;
}

