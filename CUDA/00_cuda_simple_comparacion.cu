
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sumParallel(int *dev_sum, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid <= num)
    {

	/* se usa la función atomicAdd para realizar la sumatoria y  garantizar que varios hilos no vayan a escribir simultáneamente en la misma ubicación de memoria. */
        atomicAdd(dev_sum, (int)tid);
        tid += stride; //coge el identificado unico de cada hilo y le agrega el paso para pasar al siguiente hilo
    }
}

int cuda_suma_simple(int num)
{
    int sum = 0;

    int *dev_sum; //referencia de la variable que se ubicara en el device

  
    hipMalloc((void**)&dev_sum, sizeof(int)); //reservamos espacio de memoria
    hipMemcpy(dev_sum, &sum, sizeof(int), hipMemcpyHostToDevice); //copaimos la variable desde el host al sum

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    sumParallel<<<gridSize, blockSize>>>(dev_sum, num); //invocamos el kernel sumParallel que es el que se encarga de realizar la suma
    
    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost); //copiamos el resultado ahora en sentido contrario, es decir desde el device hasta el host
    hipFree(dev_sum); //liberamos la memoria reservada

    return sum;
}
int sec_suma_simple(int num){
    int resultado = 0;
    for(int i=0; i<=num; i++){
         resultado+=i;
    }
    return resultado;
}
float run_cuda_suma_simple(int num){

     /*Inicializamos las variables con las cuales tomaremos el tiempo */
    hipEvent_t start, stop;
    float elapsedTime;
 
    // Toma el tiempo del acelerado por gpu
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); //comienza a tomar el tiempo

    int result = cuda_suma_simple(num);
    printf("Resultado Cuda = %d\n", result);

    hipEventRecord(stop, 0); //para de tomar el tiempo
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

float run_sec_suma_simple(int num){
 
     /*Inicializamos las variables con las cuales tomaremos el tiempo */
    hipEvent_t start, stop;
    float elapsedTime;
 
    // Toma el tiempo del acelerado por gpu
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0); //comienza a tomar el tiempo


    int result = sec_suma_simple(num);
    printf("Resultado Secu = %d\n", result);

    hipEventRecord(stop, 0); //para de tomar el tiempo
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}   

int main(){
    float run_time_sec, run_time_cuda;
    int num = 0;
    for (int i=0; i<=10; i++){
        printf("\n\n");
        num = pow(10, i);
        run_time_cuda = run_cuda_suma_simple(num);
        run_time_sec = run_sec_suma_simple(num);

        printf("Comparacion para n = %d\n", num);	
        printf("Tiempo Cuda (milisegundos)       = %f\n", run_time_cuda); 
        printf("Tiempo Secuencial (milisegundos) = %f\n", run_time_sec); 


        int blockSize = 256;
        int gridSize = (num + blockSize - 1) / blockSize;
        int numThreads = gridSize * blockSize;
        double speedup = run_time_cuda / run_time_sec;
        double scalability = run_time_sec / (run_time_cuda * numThreads);

        printf("Speedup: %.2f\n", speedup);
        printf("Escalabilidad: %.2f\n", scalability);

        int numOperations = num - 1;  // Número de operaciones de punto flotante realizadas
        double performance = numOperations / (run_time_cuda * 1e6);  // Rendimiento computacional en FLOPS
        printf("Performance: %.2f FLOPS\n", performance);
  
  
   }

}
